#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#define BSIZE 256

// VALORES A SUMAR SON A[] = [0,1,2,3,4,5,...., n-1]
__global__ void kernel_initarray(float *a, long n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n){
        a[tid] = tid;
    }
}

__global__ void kernel_reduction(float *a, long n){
    // AQUI PROGRAMAR SOLUCION GPU DE REDUCCION
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int half = (n+2-1)/2;
    if(tid < half){
        if(tid + half < n){
            a[tid] = a[tid] + a[tid + half];
        }
    }
}

float cpu_reduction(float *a, long n){
    float sum = 0.0f;
    // averiguen como transformar rapidamente esto en paralelo con OpenMP
    #pragma omp parallel for reduction(+:sum)
    for(long i=0; i<n; ++i){
        sum += a[i];
    }
    return sum;
}

void print_gpu_array(float *xd, long n, const char *msg){
    float *xh = new float[n];
    hipMemcpy(xh, xd, sizeof(int)*n, hipMemcpyDeviceToHost);
    printf("%s\n", msg);
    for(int i=0; i<n; ++i){
        printf("%f ", xh[i]);
    }
    printf("\n");
}

int main(int argc, char **argv){
    if(argc != 3){
        fprintf(stderr, "run as ./prog n nt\n");
        exit(EXIT_FAILURE);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    long n = atoi(argv[1]);
    int nt = atoi(argv[2]);
    omp_set_num_threads(nt);
    float *xd, *xh;
    float gpures, cpures;
    float gputime, cputime;
    dim3 block(BSIZE, 1, 1);
    dim3 grid((n+BSIZE-1)/(BSIZE), 1, 1);
    xh = new float[n];
    hipMalloc(&xd, sizeof(float)*n);

    // (1) parallel GPU init
    printf("GPU Init Array......................."); fflush(stdout);
    kernel_initarray<<<grid, block >>> (xd, n);
    hipDeviceSynchronize();
    printf("done\n"); fflush(stdout);


    // (2) parallel reduction
    int naux=n, cont = 0;
    printf("GPU reduction........................"); fflush(stdout);
    hipEventRecord(start);
    while(naux > 1){
        //printf("\n\nPASS %i, naux=%i\n", ++cont, naux);
        //print_gpu_array(xd, naux, "input");
        kernel_reduction<<<grid, block>>>(xd, naux);
        hipDeviceSynchronize();
        naux = (naux+1)/2;
        //print_gpu_array(xd, naux, "result");
    }
    printf("done\n"); fflush(stdout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gputime, start, stop);


    // (3) get result
    hipMemcpy(&gpures, xd, sizeof(float), hipMemcpyDeviceToHost);

    // (4) CPU result for validation
    printf("Init array and copy GPU -> CPU......."); fflush(stdout);
    kernel_initarray<<<grid, block >>> (xd, n);
    hipMemcpy(xh, xd, sizeof(float)*n, hipMemcpyDeviceToHost);
    printf("done\n"); fflush(stdout);
    printf("CPU reduction (%i threads)..........", nt); fflush(stdout);
    hipEventRecord(start);
    cpures = cpu_reduction(xh, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cputime, start, stop);
    printf("done\n"); fflush(stdout);
    printf("GPU result: %f  (%f secs)\nCPU result: %f  (%f secs)\n", gpures, gputime*0.001f, cpures, cputime*0.001f);
}
