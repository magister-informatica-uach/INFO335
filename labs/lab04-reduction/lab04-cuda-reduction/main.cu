#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BSIZE 256

__global__ void kernel_initarray(float *a, long n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n){
        a[tid] = tid;
    }
}

__global__ void kernel_reduction(float *a, long n){
	// (1) pasar datos de global a local
	__shared__ float sumas[BSIZE];
	// id local	
	int ltid = threadIdx.x;
	// id global
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	sumas[ltid] = 0.0f;
	if(tid < n){
		sumas[ltid] = a[tid];
	}
	__syncthreads();
	// (2) reducir por bloque en memoria compartida
	int l = BSIZE >> 1;
	while( l > 0 ){
		if(ltid < l){
			sumas[ltid] += sumas[ltid + l];
		}
		l = l >> 1;
		__syncthreads();
	}	
	// (3) reduccion global
	// a) escribir en un resultado unico por bloque, e iterar kernel	
	// a[blockIdx.x] = suma[0];
	// b) sumar todos los resultados en a[0], con operaciones atomicas.
	if(threadIdx.x == 0){
		atomicAdd(&a[0], sumas[0]);
	}
}

float cpu_reduction(float *a, long n){
    float sum = 0.0f;
    for(long i=0; i<n; ++i){
        sum += a[i];
    }
    return sum;
}

int main(int argc, char **argv){
    if(argc != 2){
        fprintf(stderr, "run as ./prog n\n");
        exit(EXIT_FAILURE);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    long n = atoi(argv[1]);
    float *xd, *xh;
    float gpures, cpures;
    float gputime, cputime;
    dim3 block(BSIZE, 1, 1);
    dim3 grid((n+BSIZE-1)/(BSIZE), 1, 1);
    xh = new float[n];
    hipMalloc(&xd, sizeof(float)*n);

    // (1) parallel GPU init
    printf("GPU Init Array......................."); fflush(stdout);
    kernel_initarray<<<grid, block >>> (xd, n);
    hipDeviceSynchronize();
    printf("done\n"); fflush(stdout);


    // (2) parallel reduction
    printf("GPU reduction........................"); fflush(stdout);
    hipEventRecord(start);
    kernel_reduction<<<grid, block>>>(xd, n);
    hipDeviceSynchronize();
    printf("done\n"); fflush(stdout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gputime, start, stop);


    // (3) get result
    hipMemcpy(&gpures, xd, sizeof(float), hipMemcpyDeviceToHost);

    // (4) CPU result for validation
    printf("Init array and copy GPU -> CPU......."); fflush(stdout);
    kernel_initarray<<<grid, block >>> (xd, n);
    hipMemcpy(xh, xd, sizeof(float)*n, hipMemcpyDeviceToHost);
    printf("done\n"); fflush(stdout);
    printf("CPU reduction........................"); fflush(stdout);
    hipEventRecord(start);
    cpures = cpu_reduction(xh, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cputime, start, stop);
    printf("done\n"); fflush(stdout);
    printf("GPU result: %f  (%f secs)\nCPU result: %f  (%f secs)\n", gpures, 
		    gputime*0.001f, cpures, cputime*0.001f);
}
