#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

// (1) haga un programa saxpy en CUDA:
//      a) funcion saxpy con mapeo de threads intercalado usando n threads (1 thread por dato).
// (2) Grafique tiempo vs n.
// (3) Reporte conclusiones sobre el rendimiento obtenido.
// (4) Compare el rendimiento vs Saxpy OpenMP, cual es mas rapida? por cuanto?


// CUDA:  sintaxis c++ con adicion de palabras claves para GPU
// CUDA = C++ + extensiones

// CUDA tiene 2 ambientes que interactuan, Host y Device
// Host: CPU (host) + RAM (en general todo lo que no es la GPU)
// Device: GPU (tiene su propio espacio de memoria, lo que esta en RAM, no esta
// automaticamente en GPU)

// WORKFLOW:
// Host trabajo previo
// Host -> Device
//         Device trabaja (ejecucion de codigo GPU -> Kernel)
// Host <- Device
// Host continue...

// Kernel
__global__ void mikernel(float a, float *x, float *y, float *s, int n){
    // SAXPY en GPU 
    // instrucciones GPU
    // STRATEGIA: 
    // a) calcular id del thread
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    // b) procesar "el" trabajo por thread
    if(tid < n){
        s[tid] = a*x[tid] + y[tid];
    }
}

void imprime(float *a, int n);
void cpu(float a, float *x, float *y, float *z, int n);

void init_vec(float *a, int n, float c){
	#pragma omp parallel for num_threads(12)
	for(int i=0; i<n; ++i){
		a[i] = c*i;
	}
}

void print_vec(float *a, int n, const char *msg){
    if(n > 32){ return; }
    printf("%s\n[", msg);
    for(int i=0; i<n; ++i){
        printf("%f ", a[i]);
    }
    printf("]\n");
}

int main(int argc,char **argv){
	if(argc != 4){
		fprintf(stderr, "error ejecutar como ./prog n mode BSIZE\n");
		exit(EXIT_FAILURE);
	}
    int n, m, bs;
    float a = 1.0f, *x,  *y,  *s;
    float   *dx, *dy, *ds;
    // obtener argumentos
    n = atoi(argv[1]);
    m = atoi(argv[2]);
    bs = atoi(argv[3]);

    // inicializar arreglos en Host (CPU)
    x = new float[n];
    y = new float[n];
    s = new float[n];
    init_vec(x, n, 1);
    print_vec(x, n, "vector x");
    init_vec(y, n, 2);
    print_vec(y, n, "vector y");
    init_vec(s, n, 0);

    // allocar memoria en device  (GPU)
    // cudaMalloc( puntero del puntero, bytes)
    hipMalloc(&dx, sizeof(float) * n);
    hipMalloc(&dy, sizeof(float) * n);
    hipMalloc(&ds, sizeof(float) * n);

    // copiar de Host -> Device
    //cudaMemcpy( destino, origen, bytes, direccion )
    hipMemcpy(dx, x, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(float)*n, hipMemcpyHostToDevice);
    //cudaMemcpy(ds, s, sizeof(float)*n, cudaMemcpyHostToDevice)

    // block -> maximo un total de 1024 threads en un bloque.
    dim3 block(bs, 1, 1);
    // grid -->  block ->   threads
    // grid 1D (porque el vector es lineal)
    // el grid esta definido en numero de bloques para x, y, z
    dim3 grid((n + bs -1)/bs, 1, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	printf("calculando...."); fflush(stdout);
	hipEventRecord(start);
	if(m){
		printf("GPU\n"); fflush(stdout);
		mikernel<<<grid, block>>>(a, dx, dy, ds, n);
	}
	else{
		printf("CPU\n"); fflush(stdout);
		cpu(a, x, y, s, n);	
	}
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if(m){
		hipMemcpy(s, ds, sizeof(float)*n, hipMemcpyDeviceToHost);
	}
	// calculo en GPU
	printf("ok: %f secs\n", milliseconds/1000.0f); fflush(stdout);
	print_vec(s, n, "vector S");		
}



void cpu(float a, float *x, float *y, float *s, int n){
	for(int i=0;i<n;i++){
		s[i]=a*x[i]+y[i];
	}
}
